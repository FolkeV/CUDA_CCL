/* MIT License
 *
 * Copyright (c) 2019 - Folke Vesterlund
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */
#include <iostream>

#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc.hpp>

#include "CCL.cuh"
#include "utils.hpp"
#include "timer.h"

int main(int argc,char **argv){
	std::string fileName;
	size_t numPixels, numRows, numCols;

	if (argc < 2){
		std::cout << "Usage: "<< argv[0] << " <image file>" << std::endl;
		return(-1);
	}
	fileName = argv[1];

	// Read image
	cv::Mat image;
	image = cv::imread(fileName, CV_LOAD_IMAGE_GRAYSCALE);
	if(!image.data){
		std::cerr << "Couldn't open file" << std::endl;
		return(-1);
	}

	if(!image.isContinuous()){
		std::cerr << "Image is not allocated with continuous data. Exiting..." << std::endl;
		return(-1);
	}
	numCols = image.cols;
	numRows = image.rows;
	numPixels = numRows*numCols;

	// Allocate GPU data
	// Uses managed data, so no explicit copies are needed
	unsigned char* d_img;
	unsigned  int* d_labels;
	hipMallocManaged(&d_labels, numPixels * sizeof(int ));
	hipMallocManaged(&d_img   , numPixels * sizeof(char));

	// Pre process image
	int imgMean = util::mean(image.data, numPixels);
	util::threshold(d_img, image.data, imgMean, numPixels);

	// Run and time kernel
	GpuTimer timer;
	timer.Start();
	connectedComponentLabeling(d_labels, d_img, numCols, numRows);
	timer.Stop();
	std::cout << "GPU code ran in: " << timer.Elapsed() << "ms" << std::endl;
//	hipDeviceSynchronize();	// Timer has syncronization built in
	
	// Count components
	unsigned int components = util::countComponents(d_labels, numPixels);
	std::cout << "Number of components: " << components << std::endl;

	// Plot result
	cv::Mat finalImage = util::postProc(d_labels, numCols, numRows);
	cv::imshow("Labelled image", finalImage);
	cv::waitKey();

	// Free memory
	hipFree(d_img);
	hipFree(d_labels);
}
